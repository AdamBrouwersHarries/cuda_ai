#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include "utility.h"
#include <hiprand/hiprand_kernel.h>


void random_initialise_state_list(int* state_list, int city_count, int state_count)
{
	for(int i = 0;i<state_count;i++)
	{
		//get the current state (pointer to the start of the state)
		int* state = &(state_list[i*city_count]);
		//initialise the state (guaranteed to be a correct tour)
		for(int j = 0;j<city_count;j++)
		{
			state[j] = j;
		}
		//uses fisher-yates shuffle
		//from: http://stackoverflow.com/a/375407
		//and: http://en.wikipedia.org/wiki/Fisher-Yates_shuffle
		int n = city_count;
		while(n>1)
		{
			int k = rand()%n;
			n--;
			int temp = state[n];
			state[n] = state[k];
			state[k] = temp;
		}	
	}
}
/*
 * prints a state in the format:
 * length : city order
 */
void print_state(int* s, int l, int city_count)
{
	printf("Length %7d :", l);
	int i = 0;
	for(i = 0;i<city_count;i++)
	{
		printf("%2d,",s[i]);
	}
	printf("\b\n");
}
/*
 * Gets the cost of an int* state using distances*
 */
int get_state_cost(int* state, int* distances, int city_count)
{
	int cost = 0;
	for(int i = 0;i<city_count;i++)
	{
		cost = cost + distances[XYW2D(state[i], state[(i+1)%city_count], city_count)];
	}
	return cost;
}
/*void print_stewart_state(state* s, int city_count)
{
	printf("TOURSIZE = %d,\n",city_count);
	printf("LENGTH = %d,\n", s->length);
	int i = 0;
	for(i = 0;i<city_count;i++)
	{
		printf("%d",(s->city_order[i])+1);
		if(i<city_count-1)
		{
			printf(",");
		}
	}
	printf("\n");
}*/
__global__ void sim_anneal(int* a_memory, int* b_memory, int* best_states, int* distances, int city_count, int iterations, hiprandState *globalRandState)
{
	int s_id = (blockIdx.x*blockDim.x) + threadIdx.x;
	
	//set up memory to hold states and lengths of states
	int *current_state, *next_state, *best_state, current_cost, next_cost, best_cost;
	//define them
	current_state = &(a_memory[s_id*city_count]);
	next_state = &(b_memory[s_id*city_count]);
	best_state = &(best_states[s_id*city_count]);
	//copy input state into current_state to initialise and into best_state[i*city_count]
	for(int i=0;i<city_count;i++)
	{
		best_state[i] = current_state[i];
	}
	//get the cost of the initial state, and therefore the current best_length
	current_cost = 0;
	for(int i = 0;i<city_count;i++)
	{
		current_cost = current_cost + distances[XYW2D(current_state[i], current_state[(i+1)%city_count], city_count)];
	}
	best_cost = current_cost;
	float t;//current temperature
	int k = iterations; int kmax = iterations;
	for(int i = 0;i<iterations;i++)
	{
		t = ((float)(k))/((float)(kmax));
		//first reverse a subset of the current state into the next state
		//get the start and end of the subset
		int start = (int)(hiprand_uniform(&(globalRandState[s_id]))*(float)(city_count-1)); //start in range 0->(city_count-2)
		int end = start+(int)(hiprand_uniform(&(globalRandState[s_id]))*(float)(city_count-start)); //end in range start->(city_count-1)
		//copy in data before reversed section
		for(int i = 0;i<start;i++)
		{
			next_state[i] = current_state[i];
		}
		//copy in data after reversed section
		for(int i = end+1;i<city_count;i++)
		{
			next_state[i] = current_state[i];
		}
		//reverse copy section		
		for(;start<=end;start++,end--)
		{
			next_state[start] = current_state[end];
			next_state[end] = current_state[start];
		}
		//get the length of the new state we've made
		next_cost = 0;
		for(int i = 0;i<city_count;i++)
		{
			next_cost = next_cost + distances[XYW2D(next_state[i], next_state[(i+1)%city_count], city_count)];
		}
		//switch them if the new node is shorter
		if(next_cost<current_cost)
		{
			//but first check to see if it's the best we've found
			if(next_cost<best_cost)
			{
				//if it is, assign the costs, and copy the state over
				best_cost = next_cost;
				//however as we're only doing hill climbing for now - leave it commented for efficency
				for(int i= 0;i<city_count;i++)
				{
					best_state[i]=next_state[i];				
				}
			}
			//swap the pointers
			int *temp_ptr = current_state;
			current_state = next_state;
			next_state = temp_ptr;
			//swap the costs (don't need to put the cur into next, will be recalculated)
			current_cost = next_cost;
		}else{
			//check to see what the temperature says, and weather we'll copy anyway
			float rn = hiprand_uniform(&(globalRandState[s_id]));
			float acc_prob = 1/exp(abs(current_cost-next_cost)/t);
			if(rn>acc_prob)
			{
				//swap anyway
				//swap the pointers
				int *temp_ptr = current_state;
				current_state = next_state;
				next_state = temp_ptr;
				//swap the costs (don't need to put the cur into next, will be recalculated)
				current_cost = next_cost;
			}
		}	
	}
}
__global__ void setup_kernel_randomness(hiprandState * state, unsigned long seed)
{
	int s_id = (blockIdx.x*blockDim.x) + threadIdx.x;
	hiprand_init(seed*s_id, s_id, 0, &state[s_id]);
}

int main(int argc, char** argv)
{
	printf("Format: cuda_tsp <infile> <iterations> <blockCount> <threadCount>\n");
	int iterations = atoi(argv[2]);
	int blockCount = atoi(argv[3]);
	int threadCount = atoi(argv[4]);
	printf("Iterations: %d\nblockCount: %d\nthreadCount: %d\n", iterations, blockCount, threadCount);
	printf("Started\n");
	/* set up citygraph stuff */
	srand(time(NULL));
	//read in the file to a c_string
	char* file_data = file_to_cstring(argv[1]);
	//printf("data in the file:\n%s",file_data);
	//create a char** array to hold the tokens in the string
	char** token_array = 0;
	//split the string into tokens, get the number of tokens
	cstring_to_token_array(file_data,",\r\n= ",&token_array);
	printf("Read token array\n");
	//get the city distances
	int* h_city_distances;
	int city_count = token_array_to_graph(token_array, &h_city_distances);
	printf("Created graph\n");
	//allocate device space for city distances
	int* d_city_distances;
	hipMalloc(&d_city_distances, city_count*city_count*sizeof(int));
	//copy city distances to the device
	hipMemcpy(d_city_distances, h_city_distances, city_count*city_count*sizeof(int), hipMemcpyHostToDevice);
	/*sort out memory and stuff*/
	int state_count = blockCount*threadCount;
	//initialise list of states to copy to the cuda memory
	int *h_state_list, *d_a_mem, *d_b_mem, *d_best_states;
	//initialise host space for states
	h_state_list = (int*)malloc(state_count*city_count*sizeof(int));
	//initialise device space for states
	hipMalloc(&d_a_mem, state_count*city_count*sizeof(int));
	hipMalloc(&d_b_mem, state_count*city_count*sizeof(int));
	hipMalloc(&d_best_states, state_count*city_count*sizeof(int));
	printf("All space allocated\n");
	//set up initial values for states
	random_initialise_state_list(h_state_list, city_count, state_count);
	printf("States initialised\n");
	//copy initialised states to device (dst, src)
	hipMemcpy(d_a_mem, h_state_list, state_count*city_count*sizeof(int), hipMemcpyHostToDevice);
	
	printf("Starting CUDA code\n");
	//before sim_annealing, set up the random numbers
	hiprandState* devStates;
	hipMalloc(&devStates, state_count*sizeof(hiprandState));
	setup_kernel_randomness<<<(state_count+255)/256, 256>>>(devStates, time(NULL));
	sim_anneal<<<blockCount, threadCount>>>(d_a_mem, d_b_mem, d_best_states, d_city_distances, city_count, iterations, devStates);
	printf("Finished\n");
	//copy the calculated states back
	hipMemcpy(h_state_list, d_best_states, state_count*city_count*sizeof(int), hipMemcpyDeviceToHost);
	//iterate over the list of calculated states, and find the best - serial
	int* current_best_state;
	int best_length = INT_MAX;
	for(int i = 0;i<state_count;i++)
	{
		int* current_state = &(h_state_list[i*city_count]);
		int new_length = get_state_cost(current_state, h_city_distances, city_count);
		//print_state(current_state, new_length, city_count);
		//check if better, etc...
		if(new_length < best_length)
		{
			printf("LF: %8d\n", new_length);
			current_best_state = current_state;
			best_length = new_length;
		}
	}
	printf("best state = ");print_state(current_best_state, best_length, city_count);
}